// Salt and pepper noise simulation with Cuda C/C++
// Original framework for code taken from imflipG.cu
// Modified by Ethan Webster

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <iostream>
#include <math.h>
#include <ctype.h>

#include <time.h>

#define DATAMB(bytes)			(bytes/1024/1024)
#define DATABW(bytes,timems)	((float)bytes/(timems * 1.024*1024.0*1024.0))
#define	CEIL(a,b)				((a+b-1)/b)

typedef unsigned char uch;
typedef unsigned long ul;
typedef unsigned int  ui;

uch *TheImg, *CopyImg;					// Where images are stored in CPU
uch *GPUImg, *GPUCopyImg, *GPUptr, *GPUResult;	// Where images are stored in GPU


struct ImgProp{
	int Hpixels;
	int Vpixels;
	uch HeaderInfo[54];
	ul Hbytes;
} ip;


#define	IPHB		ip.Hbytes
#define	IPH			ip.Hpixels
#define	IPV			ip.Vpixels
#define	IMAGESIZE	(IPHB*IPV)
#define	IMAGEPIX	(IPH*IPV)


// Kernel that adds salt&pepper noise of given probability density to an image
__global__
void corruptPixels(uch *ImgDst, uch *ImgSrc, ui Hpixels, ui Vpixels, double prob)
{
	// threads/blocks info and IDs
	ui ThrPerBlk = blockDim.x;
	ui MYbid = blockIdx.x;
	ui MYtid = threadIdx.x;
	ui MYgtid = ThrPerBlk * MYbid + MYtid;

	//ui NumBlocks = gridDim.x;
	ui BlkPerRow = CEIL(Hpixels, ThrPerBlk);
	ui MYrow = MYbid / BlkPerRow;
	ui MYcol = MYgtid - MYrow*BlkPerRow*ThrPerBlk;

	// leave buffer frame around image to avoid 8 edge cases for convolutions
	if (MYcol > Hpixels-4 || MYcol < 3 || MYrow > Vpixels-4 || MYrow < 3) return;

	ui MYpixIndex = MYrow * Hpixels + MYcol; // pixel index in B&W image

	ui RowBytes = (Hpixels * 3 + 3) & (~3);  // bytes in row of R=G=B grayscale output image

	ui MYresultIndex = MYrow * RowBytes + 3 * MYcol; // pixel index in grayscale image (R=B=G)

	// seed cuRAND random number generator function with clock cycle + threadID
	hiprandState state;
	hiprand_init((unsigned long long)clock() + MYtid, 0, 0, &state);

	// sample uniform distribution from 0 to 255 (random pixel intensity)
	ui loc = ((ui)(hiprand(&state)))%255;

/* 
	half of the probability is used for the following because
	the salt vs pepper contribution is split 50/50 
*/
	// if pixel intensity is located in the lower half of the 
	// probability region, then add pepper noise
	if( loc <= (ui)(prob/2.0f)) {
		ImgDst[MYresultIndex] = 0;
		ImgDst[MYresultIndex+1] = 0;
		ImgDst[MYresultIndex+2] = 0;
	}

	// otherwise if pixel intensity is located in the upper half of the 
	// probability region, then add salt noise
	else if(loc > (ui)(prob/2.0f) && loc < (ui)prob ) {
		ImgDst[MYresultIndex] = 255;
		ImgDst[MYresultIndex+1] = 255;
		ImgDst[MYresultIndex+2] = 255;
	}

	// if we reached this, then no noise is added
	else {
		ImgDst[MYresultIndex] = ImgSrc[MYpixIndex];
		ImgDst[MYresultIndex+1] = ImgSrc[MYpixIndex];
		ImgDst[MYresultIndex+2] = ImgSrc[MYpixIndex];
	}


}


// Kernel that calculates a B&W image from an RGB image
// resulting image has a double type for each pixel position
__global__
void BWKernel(uch *ImgBW, uch *ImgGPU, ui Hpixels)
{
	ui ThrPerBlk = blockDim.x;
	ui MYbid = blockIdx.x;
	ui MYtid = threadIdx.x;
	ui MYgtid = ThrPerBlk * MYbid + MYtid;
	double R, G, B;

	//ui NumBlocks = gridDim.x;
	ui BlkPerRow = CEIL(Hpixels, ThrPerBlk);
	ui RowBytes = (Hpixels * 3 + 3) & (~3);
	ui MYrow = MYbid / BlkPerRow;
	ui MYcol = MYgtid - MYrow*BlkPerRow*ThrPerBlk;
	if (MYcol >= Hpixels) return;			// col out of range

	ui MYsrcIndex = MYrow * RowBytes + 3 * MYcol;
	ui MYpixIndex = MYrow * Hpixels + MYcol;

	B = (double)ImgGPU[MYsrcIndex];
	G = (double)ImgGPU[MYsrcIndex + 1];
	R = (double)ImgGPU[MYsrcIndex + 2];
	ImgBW[MYpixIndex] = (uch)((R+G+B)/3.0);
}


// Kernel that copies an image from one part of the
// GPU memory (ImgSrc) to another (ImgDst)
__global__
void PixCopy(uch *ImgDst, uch *ImgSrc, ui FS)
{
	ui ThrPerBlk = blockDim.x;
	ui MYbid = blockIdx.x;
	ui MYtid = threadIdx.x;
	ui MYgtid = ThrPerBlk * MYbid + MYtid;

	if (MYgtid > FS) return;				// outside the allocated memory
	ImgDst[MYgtid] = ImgSrc[MYgtid];
}



// Read a 24-bit/pixel BMP file into a 1D linear array.
// Allocate memory to store the 1D image and return its pointer.
uch *ReadBMPlin(char* fn)
{
	static uch *Img;
	FILE* f = fopen(fn, "rb");
	if (f == NULL){	printf("\n\n%s NOT FOUND\n\n", fn);	exit(EXIT_FAILURE); }

	uch HeaderInfo[54];
	fread(HeaderInfo, sizeof(uch), 54, f); // read the 54-byte header
	// extract image height and width from header
	int width = *(int*)&HeaderInfo[18];			ip.Hpixels = width;
	int height = *(int*)&HeaderInfo[22];		ip.Vpixels = height;
	int RowBytes = (width * 3 + 3) & (~3);		ip.Hbytes = RowBytes;
	//save header for re-use
	memcpy(ip.HeaderInfo, HeaderInfo,54);
	printf("\n Input File name: %17s  (%u x %u)   File Size=%u", fn, 
			ip.Hpixels, ip.Vpixels, IMAGESIZE);
	// allocate memory to store the main image (1 Dimensional array)
	Img  = (uch *)malloc(IMAGESIZE);
	if (Img == NULL) return Img;      // Cannot allocate memory
	// read the image from disk
	fread(Img, sizeof(uch), IMAGESIZE, f);
	fclose(f);
	return Img;
}


// Write the 1D linear-memory stored image into file.
void WriteBMPlin(uch *Img, char* fn)
{
	FILE* f = fopen(fn, "wb");
	if (f == NULL){ printf("\n\nFILE CREATION ERROR: %s\n\n", fn); exit(1); }
	//write header
	fwrite(ip.HeaderInfo, sizeof(uch), 54, f);
	//write data
	fwrite(Img, sizeof(uch), IMAGESIZE, f);
	printf("\nOutput File name: %17s  (%u x %u)   File Size=%u", fn, ip.Hpixels, ip.Vpixels, IMAGESIZE);
	fclose(f);
}


int main(int argc, char **argv)
{

	hipError_t cudaStatus;
	char InputFileName[255], OutputFileName[255], ProgName[255];
	ui BlkPerRow, ThrPerBlk=256, NumBlocks;
	//cudaDeviceProp GPUprop;
	//ul SupportedKBlocks, SupportedMBlocks, MaxThrPerBlk;		char SupportedBlocks[100]; 

	ui amt, GPUtotalBufferSize;

	double inputNL;

	strcpy(ProgName, "randNoise");
	switch (argc){
	case 5:  ThrPerBlk=atoi(argv[4]);
	case 4:  amt=atoi(argv[3]);
	case 3:  strcpy(InputFileName, argv[1]);
			 strcpy(OutputFileName, argv[2]);
			 break;
	default: printf("\n\nUsage:   %s InputFilename OutputFilename [NoiseDensity] [ThrPerBlk]", ProgName);
			 printf("\n\nExample: %s Astronaut.bmp Output.bmp", ProgName);
			 printf("\n\nExample: %s Astronaut.bmp Output.bmp 50", ProgName);
			 printf("\n\nExample: %s Astronaut.bmp Output.bmp 50  128",ProgName);
			 printf("\n\nNoise Density is in percent, from 0-100\n\n");
			 exit(EXIT_FAILURE);
	}
	if (amt > 100) {
		printf("Invalid noise amount. Must be between 0 and 100");
		exit(EXIT_FAILURE);
	}
	if ((ThrPerBlk < 32) || (ThrPerBlk > 1024)) {
		printf("Invalid ThrPerBlk option '%u'. Must be between 32 and 1024. \n", ThrPerBlk);
		exit(EXIT_FAILURE);
	}

	// Create CPU memory to store the input and output images
	TheImg = ReadBMPlin(InputFileName); // Read the input image if memory can be allocated
	if (TheImg == NULL){
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}
	CopyImg = (uch *)malloc(IMAGESIZE);
	if (CopyImg == NULL){
		free(TheImg);
		printf("Cannot allocate memory for the input image...\n");
		exit(EXIT_FAILURE);
	}


	// Choose which GPU to run on, change this on a multi-GPU system.
	int NumGPUs = 0;
	hipGetDeviceCount(&NumGPUs);
	if (NumGPUs == 0){
		printf("\nNo CUDA Device is available\n");
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		exit(EXIT_FAILURE);
	}

	
	// cudaGetDeviceProperties(&GPUprop, 0);
	// SupportedKBlocks = (ui)GPUprop.maxGridSize[0] * (ui)GPUprop.maxGridSize[1] * (ui)GPUprop.maxGridSize[2] / 1024;
	// SupportedMBlocks = SupportedKBlocks / 1024;
	// sprintf(SupportedBlocks, "%u %c", (SupportedMBlocks >= 5) ? SupportedMBlocks : SupportedKBlocks, (SupportedMBlocks >= 5) ? 'M' : 'K');
	// MaxThrPerBlk = (ui)GPUprop.maxThreadsPerBlock;

	// allocate sufficient memory on the GPU to hold B&W image and grayscale output image
	GPUtotalBufferSize = IMAGEPIX+IMAGESIZE;
	cudaStatus = hipMalloc((void**)&GPUptr, GPUtotalBufferSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed! Can't allocate GPU memory");
		exit(EXIT_FAILURE);
	}

	// setup pointers to B&W image and output corrupted image
	GPUImg			= (uch *)GPUptr;
	GPUCopyImg	= GPUImg + IMAGESIZE;

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(GPUImg, TheImg, IMAGESIZE, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy  CPU to GPU  failed!");
		exit(EXIT_FAILURE);
	}

	BlkPerRow = CEIL(ip.Hpixels, ThrPerBlk);
	NumBlocks = IPV*BlkPerRow;

	BWKernel <<< NumBlocks, ThrPerBlk >>> (GPUCopyImg, GPUImg, IPH);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n\n hipDeviceSynchronize 1 returned error code %d after launching the kernel!\n", cudaStatus);
		exit(EXIT_FAILURE);
	}

	// add random noise to the image
	inputNL = 255.0f*(double)amt/100.0f;
		 corruptPixels <<< NumBlocks, ThrPerBlk >>> (GPUImg, GPUCopyImg, IPH, IPV, inputNL);
				  GPUResult = GPUImg;


	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n\nhipDeviceSynchronize 2 returned error code %d after launching the kernel!\n", cudaStatus);
		exit(EXIT_FAILURE);
	}
	

	// Copy output (results) from GPU buffer to host (CPU) memory.
	cudaStatus = hipMemcpy(CopyImg, GPUResult, IMAGESIZE, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy GPU to CPU  failed!");
		exit(EXIT_FAILURE);
	}
	

	cudaStatus = hipDeviceSynchronize();
	//checkError(cudaGetLastError());	// screen for errors in kernel launches
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n Program failed after cudaDeviceSynchronize()!");
		free(TheImg);
		free(CopyImg);
		exit(EXIT_FAILURE);
	}


	WriteBMPlin(CopyImg, OutputFileName);		// Write the flipped image back to disk

	printf("\n\n--------------------------------------------------------------------------\n");
	printf("Successfully added %d%% noise to the given image and converted to grayscale.\n", amt);

	// Deallocate CPU, GPU memory and destroy events.
	hipFree(GPUptr);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		free(TheImg);
		free(CopyImg);
		exit(EXIT_FAILURE);
	}

	free(TheImg);
	free(CopyImg);
	return(EXIT_SUCCESS);
}



